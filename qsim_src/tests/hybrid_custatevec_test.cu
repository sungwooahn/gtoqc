// Copyright 2019 Google LLC. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     https://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "hybrid_testfixture.h"

#include <hipblas.h>
#include <custatevec.h>

#include "gtest/gtest.h"

#include "../lib/simulator_custatevec.h"

namespace qsim {

template <typename FP>
struct Factory {
  using fp_type = FP;
  using Simulator = qsim::SimulatorCuStateVec<fp_type>;
  using StateSpace = typename Simulator::StateSpace;

  Factory() {
    ErrorCheck(hipblasCreate(&cublas_handle));
    ErrorCheck(custatevecCreate(&custatevec_handle));
  }

  ~Factory() {
    ErrorCheck(hipblasDestroy(cublas_handle));
    ErrorCheck(custatevecDestroy(custatevec_handle));
  }

  StateSpace CreateStateSpace() const {
    return StateSpace(cublas_handle, custatevec_handle);
  }

  Simulator CreateSimulator() const {
    return Simulator(cublas_handle, custatevec_handle);
  }

  hipblasHandle_t cublas_handle;
  custatevecHandle_t custatevec_handle;
};

TEST(HybridCuStateVecTest, Hybrid2) {
  TestHybrid2(qsim::Factory<float>());
}

TEST(HybridCuStateVecTest, Hybrid4) {
  TestHybrid4(qsim::Factory<float>());
}

}  // namespace qsim

int main(int argc, char** argv) {
  ::testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}
